#include "hip/hip_runtime.h"
#include "kernels.h"
#include "util.h"

#define get_tid() (blockDim.x * (blockIdx.y * gridDim.x + blockIdx.x) + threadIdx.x)
#define get_bid() (blockIdx.y * gridDim.x + blockIdx.x)

void CudaApp::init(int fd, int bufferSize)
{
    hipExternalMemoryHandleDesc externalMemoryDesc = {};
    externalMemoryDesc.type = hipExternalMemoryHandleTypeOpaqueFd;
    externalMemoryDesc.handle.fd = fd; // File descriptor from Vulkan
    externalMemoryDesc.size = bufferSize;

    hipExternalMemory_t cudaExternalMemory;
    hipImportExternalMemory(&cudaExternalMemory, &externalMemoryDesc);

    hipExternalMemoryBufferDesc bufferDesc = {};
    bufferDesc.offset = 0;
    bufferDesc.size = bufferSize;

    void* devPtrVoid;
    hipExternalMemoryGetMappedBuffer(&devPtrVoid, cudaExternalMemory, &bufferDesc);
    devPtr = reinterpret_cast<float*>(devPtrVoid);
}

__global__ void changeColors(float* vertexBuffer)
{
    auto tid = get_tid();
    float& val = *(vertexBuffer + tid * sizeof(Vertex) / sizeof(float) + 2);
    val += 0.004f;
    if (val > 1.0f)
        val = 0.0f;
}

void CudaApp::step()
{
    changeColors<<<1, 4>>>(devPtr);
}
