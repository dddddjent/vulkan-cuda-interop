#include "hip/hip_runtime.h"
#include "kernels.h"
#include "util.h"

#define get_tid() (blockDim.x * (blockIdx.y * gridDim.x + blockIdx.x) + threadIdx.x)
#define get_bid() (blockIdx.y * gridDim.x + blockIdx.x)

void CudaApp::init(int fd, int bufferSize)
{
    hipExternalMemoryHandleDesc externalMemoryDesc = {};
    externalMemoryDesc.type = hipExternalMemoryHandleTypeOpaqueFd;
    externalMemoryDesc.handle.fd = fd; // File descriptor from Vulkan
    externalMemoryDesc.size = bufferSize;

    hipExternalMemory_t cudaExternalMemory;
    hipImportExternalMemory(&cudaExternalMemory, &externalMemoryDesc);

    hipExternalMemoryBufferDesc bufferDesc = {};
    bufferDesc.offset = 0;
    bufferDesc.size = bufferSize;

    void* devPtrVoid;
    hipExternalMemoryGetMappedBuffer(&devPtrVoid, cudaExternalMemory, &bufferDesc);
    devPtr = reinterpret_cast<Vertex*>(devPtrVoid);
}

__global__ void changeColors(Vertex* vertexBuffer)
{
    auto tid = get_tid();
    vertexBuffer[tid].color.r += 0.004f;
    if (vertexBuffer[tid].color.r > 1.0f)
        vertexBuffer[tid].color.r = 0.0f;
}

void CudaApp::step()
{
    changeColors<<<1, 4>>>(devPtr);
}
