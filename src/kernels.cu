#include "hip/hip_runtime.h"
#include "kernels.h"
#include "util.h"

#define get_tid() (blockDim.x * (blockIdx.y * gridDim.x + blockIdx.x) + threadIdx.x)
#define get_bid() (blockIdx.y * gridDim.x + blockIdx.x)

void CudaApp::initSemaphore(int vkCudaFd, int cudaVkFd)
{
    hipExternalSemaphoreHandleDesc externalSemaphoreHandleDesc;
    memset(&externalSemaphoreHandleDesc, 0, sizeof(externalSemaphoreHandleDesc));
    externalSemaphoreHandleDesc.type = hipExternalSemaphoreHandleTypeOpaqueFd;
    externalSemaphoreHandleDesc.handle.fd = cudaVkFd;
    externalSemaphoreHandleDesc.flags = 0;
    hipImportExternalSemaphore(
        &cudaVkSemaphore,
        &externalSemaphoreHandleDesc);

    memset(&externalSemaphoreHandleDesc, 0, sizeof(externalSemaphoreHandleDesc));
    externalSemaphoreHandleDesc.type = hipExternalSemaphoreHandleTypeOpaqueFd;
    externalSemaphoreHandleDesc.handle.fd = vkCudaFd;
    externalSemaphoreHandleDesc.flags = 0;
    hipImportExternalSemaphore(
        &vkCudaSemaphore,
        &externalSemaphoreHandleDesc);
}

void CudaApp::initMemHandle(int fd, int bufferSize)
{
    hipExternalMemoryHandleDesc externalMemoryDesc = {};
    externalMemoryDesc.type = hipExternalMemoryHandleTypeOpaqueFd;
    externalMemoryDesc.handle.fd = fd; // File descriptor from Vulkan
    externalMemoryDesc.size = bufferSize;

    hipImportExternalMemory(&cudaExternalMemory, &externalMemoryDesc);

    hipExternalMemoryBufferDesc bufferDesc = {};
    bufferDesc.offset = 0;
    bufferDesc.size = bufferSize;

    void* devPtrVoid;
    hipExternalMemoryGetMappedBuffer(&devPtrVoid, cudaExternalMemory, &bufferDesc);
    devPtr = reinterpret_cast<Vertex*>(devPtrVoid);
}

__global__ void changeColors(Vertex* vertexBuffer)
{
    auto tid = get_tid();
    vertexBuffer[tid].color.r += 0.004f;
    if (vertexBuffer[tid].color.r > 1.0f)
        vertexBuffer[tid].color.r = 0.0f;
}

void CudaApp::step()
{
    changeColors<<<1, 4>>>(devPtr);
}

void CudaApp::cleanup()
{
    hipDestroyExternalMemory(cudaExternalMemory);
    hipDestroyExternalSemaphore(cudaVkSemaphore);
    hipDestroyExternalSemaphore(vkCudaSemaphore);
}
